#include "hip/hip_runtime.h"
#include <culibtest/SomeClass.h>
#include "../include/culibtest/SomeClass.h"
#include <stdio.h>
#include <cstdlib>
#include <cstring>

#include <hip/hip_runtime.h>

namespace culibtest {
	
	SomeClass::SomeClass() : 
		privateInt(0),
		publicInt(0)
	{
		
	}
	
	SomeClass::~SomeClass(){
		
	}
	
	bool SomeClass::sayHello(){
		printf("SomeClass says HelloWorld!\n");
		return true;
	}
	
	unsigned int SomeClass::getPrivateInt(){
		return this->privateInt;
	}
	
	bool SomeClass::setPrivateInt(unsigned int value){
		this->privateInt = value;
		return this->privateInt == value;
	}


	unsigned int SomeClass::launchRandomKernal(unsigned int(*device_function_ptr)(unsigned int), const unsigned int N) {
		hipError_t status;


		unsigned int * h_values = (unsigned int *)std::malloc(N * sizeof(unsigned int));
		unsigned int * d_values = nullptr;
		status = hipMalloc((void**)&d_values, N * sizeof(unsigned int));


		if (h_values == nullptr || d_values == nullptr) {
			free(h_values);
			hipFree(d_values);
			return 0;
		}
		// Reset to 0
		std::memset(h_values, 0, N * sizeof(unsigned int));
		status = hipMemset(d_values, 0, N * sizeof(unsigned int));

		// Calculate kernal launch parameters
		int minGridSize = 0;
		int blockSize = 0;
		int gridSize = 0;
		
		status = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			simple_kernal, 0, 0);
		gridSize = (N + blockSize - 1) / blockSize;


		// Call function
		simple_kernal << < gridSize, blockSize >> >(device_function_ptr, N, d_values);
		status = hipDeviceSynchronize();
		status = hipGetLastError();
		if (hipSuccess != status) {
			printf("cuda error %s:%d!\n\t%d:%s\n", __FILE__, __LINE__, status, hipGetErrorString(status));
		}


		// Copy data back to host.
		status = hipMemcpy(h_values, d_values, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

		// Accumulate
		unsigned int sum = 0;
		for (unsigned int i = 0; i < N; i++) {
			sum += h_values[i];
		}

		// Free memory

		free(h_values);
		status = hipFree(d_values);

		// Return the accumulated value.
		return sum;
	}

	// Not a class member
	__global__ void simple_kernal(unsigned int(*device_function_ptr)(unsigned int), unsigned int N, unsigned int * d_indices) {
		// Get global index
		unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

		if (idx < N) {
			
			d_indices[idx] = device_function_ptr(N);
			if (idx < 16) {
				printf("tid %u: value %u\n", idx, d_indices[idx]);
			}
		}
	}
	
}
