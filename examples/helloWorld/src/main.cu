#include <stdio.h>

#include <culibtest.h>

typedef unsigned int(*dfuncptr)(unsigned int);

// __constant__ unsigned int d_twelve = 0;

__device__ unsigned int someDeviceFunction(unsigned int N) {
	// printf("someDeviceFunction\n");
	//unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	//return idx;
	return d_twelve;
	// return 1;
}

__device__ dfuncptr someDeviceFunction_ptr = someDeviceFunction;



int main(int argc, char * argv[]){
	hipError_t status = hipSuccess;
	printf("helloWorld Example:\n");

	
	culibtest::SomeClass obj = culibtest::SomeClass();
	
	obj.sayHello();
	obj.setPrivateInt(12);
	printf("privateInt %u\n", obj.getPrivateInt());

	//someDeviceFunction_ptr = &someDeviceFunction; // bad
	dfuncptr h_someDeviceFunction_ptr;
	status = hipMemcpyFromSymbol(&h_someDeviceFunction_ptr, HIP_SYMBOL(someDeviceFunction_ptr), sizeof(dfuncptr));
	if (status != hipSuccess) {
		printf("Error, could not get devidce pointer.\n");
		return 1;
	}

	printf("%p\n", h_someDeviceFunction_ptr);
	unsigned int sum = obj.launchRandomKernal(h_someDeviceFunction_ptr, 1024);

	status = hipDeviceSynchronize();
	if (hipSuccess != status) {
		printf("cuda error %s:%d!\n\t%d:%s\n", __FILE__, __LINE__, status, hipGetErrorString(status));
	}
	status = hipGetLastError();
	if (hipSuccess != status) {
		printf("cuda error %s:%d!\n\t%d:%s\n", __FILE__, __LINE__, status, hipGetErrorString(status));
	}

	printf("sum: %u\n", sum);


	unsigned int l_twelve = 0;
	status = hipMemcpyFromSymbol(&l_twelve, HIP_SYMBOL(d_twelve), sizeof(unsigned int));
	if (hipSuccess != status) {
		printf("cuda error %s:%d!\n\t%d:%s\n", __FILE__, __LINE__, status, hipGetErrorString(status));
	}
	printf("l_twelve %u\n", l_twelve);
	
	
}
