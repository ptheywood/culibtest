#include <stdio.h>

#include <culibtest.h>

typedef unsigned int(*dfuncptr)(unsigned int);


__device__ unsigned int someDeviceFunction(unsigned int N) {
	printf("someDeviceFunction\n");
	//unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	//return idx;
	return 1;
}

__device__ dfuncptr someDeviceFunction_ptr = someDeviceFunction;



int main(int argc, char * argv[]){
	printf("helloWorld Example:\n");
	
	culibtest::SomeClass obj = culibtest::SomeClass();
	
	obj.sayHello();
	obj.setPrivateInt(12);
	printf("privateInt %u\n", obj.getPrivateInt());



	//someDeviceFunction_ptr = &someDeviceFunction; // bad
	dfuncptr h_someDeviceFunction_ptr;
	hipError_t status = hipSuccess;
	status = hipMemcpyFromSymbol(&h_someDeviceFunction_ptr, HIP_SYMBOL(someDeviceFunction_ptr), sizeof(dfuncptr));
	if (status != hipSuccess) {
		printf("Error, could not get devidce pointer.\n");
		return 1;
	}

	printf("%p, %p\n", someDeviceFunction_ptr, h_someDeviceFunction_ptr);
	unsigned int sum = obj.launchRandomKernal(h_someDeviceFunction_ptr, 1024);

	printf("sum: %u\n", sum);
	
	
}
